
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdbool.h> /*helps with bool data type*/
#include <string.h> /* memset */
#include <unistd.h> /* close */
#include <emmintrin.h>
#include <sys/time.h> /*allows system type*/





struct timeval start, end;
void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

//initializes matrix values to 0,
//AS OF RIGHT NOW , ONLY USES THREADS!!!
__global__ void initTo(int *matx, int vs, int set){
	int index = threadIdx.x;
  	int stride = blockDim.x;
	for(int i = index ; i < vs; i += stride ){
		matx[i] = set;
	}

}
//initializes truth table to true
////AS OF RIGHT NOW , ONLY USES THREADS!!!
__global__ void initTo(bool *t_table, int vs, bool set){
	int index = threadIdx.x;
  	int stride = blockDim.x;
	for(int i = index ; i < vs; i += stride ){
		t_table[i] = set;
	}

}

__global__ void checkPathsTH(int *matrix, int vertices, int pathsize, int *paths, bool *t_table){
	//int index = threadIdx.x;
  	int stride = blockDim.x;
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	//printf("%i\n",stride);
	//index is the actual index!!!
	//so 0-95
	//stops trash from going out of bounds
/*
	if(index<(pathsize)){
		for(int j = 0;j<vertices;j++){
			printf("%i's own %i called %i ",index,j,paths[(index*vertices)+j]);
			//THIS GETS YOU WHAT YOU NEED paths[(index*vertices)+j]
		}
		//printf("%i, %i, %i\n",index,stride,paths[index]);
	}
*/


	for(int i = index ; i < pathsize; i += stride){

		for(int j = 0;j<vertices;j++){

			if(j==0){
				if(matrix[(paths[(index*vertices)+j] * vertices) + paths[(index*vertices)+j+1]] != 1 &&
					matrix[(paths[(index*vertices)+j+1] * vertices) + paths[(index*vertices)+j]] != 1 )
				{
					t_table[index] = false;
					break;
				}
			}else if(j > 0 && j < vertices-1){
				if( (matrix[(paths[(index*vertices)+j] * vertices) + paths[(index*vertices)+j+1]] != 1 &&
					matrix[(paths[(index*vertices)+j+1] * vertices) + paths[(index*vertices)+j]] != 1 )
					||
				    (matrix[(paths[(index*vertices)+j] * vertices) + paths[(index*vertices)+j-1]] != 1 &&
					matrix[(paths[(index*vertices)+j-1] * vertices) + paths[(index*vertices)+j]] != 1 )
					)
					
				{
					t_table[index] = false;
					break;
				}

			}else if(j== vertices -1){
				if(matrix[(paths[(index*vertices)+j] * vertices) + paths[(index*vertices)+j-1]] != 1 &&
					matrix[(paths[(index*vertices)+j-1] * vertices) + paths[(index*vertices)+j]] != 1 )
				{
					t_table[index] = false;
					break;
				}


			}
			
		}
	}	

	//__syncthreads();
}


void checkPathLocal(int *ptr_path,int vertices,int pathsize, int *matrix, bool *possiblePaths ){
	for(int outcount = 0; outcount < pathsize;outcount++){
        for (int i = 0; i < vertices; i++){

            //the first one only checks the next one, checks if the path is even possible
            if(i == 0){
                if(matrix[(ptr_path[(outcount*vertices)+i] * vertices) + ptr_path[(outcount*vertices)+i+1]] != 1 &&
                   matrix[(ptr_path[(outcount*vertices)+i+1] * vertices) + ptr_path[(outcount*vertices)+i]] != 1)
                {
                    possiblePaths[outcount] = false;
                    break;
                }
            }
            else if(i > 0 && i < vertices - 1){
                if( (matrix[(ptr_path[(outcount*vertices)+i] * vertices) + ptr_path[(outcount*vertices)+i+1]] != 1 &&
                     matrix[(ptr_path[(outcount*vertices)+i+1] * vertices) + ptr_path[(outcount*vertices)+i]] != 1) 
                    ||
                    (matrix[(ptr_path[(outcount*vertices)+i] * vertices) + ptr_path[(outcount*vertices)+i-1]] != 1  &&
                     matrix[(ptr_path[(outcount*vertices)+i-1] * vertices) + ptr_path[(outcount*vertices)+i]] != 1)
                    )
                {
                    possiblePaths[outcount] = false;
                    break;
                }
            }
            else if(i == vertices - 1){
                if( matrix[(ptr_path[(outcount*vertices)+i] * vertices) + ptr_path[(outcount*vertices)+i-1]] != 1  &&
                    matrix[(ptr_path[(outcount*vertices)+i-1] * vertices) + ptr_path[(outcount*vertices)+i]] != 1)
                {
                    possiblePaths[outcount] = false;
                    break;
                } 
            }
        }   
    }
	endtime("NORMAL");
}




//just makes unweighted edges in a 1D array
void makeEdge(int to, int from, int *num, int vertices) {
        *(num + (((to) * (vertices)) + (from))) = 1;
	*(num + (((from) * (vertices)) + (to))) = 1;
	//matrix[((to) * (vertices)) + (from)] = 1;
        //matrix[((from) * (vertices)) + (to)] = 1;
        //matrix[from][to] = 1;
}

//swaps values, assists permute
    int swap(int a[], int i, int j)
    {
        int temp = a[i];
        a[i] = a[j];
        a[j] = temp;
        return a[j];
    }


//places in array
void getArray(int ha[] , int *ptr_path , int vertices, int pathsize){
        //finds the first path that is empty and places the array there!!!
        for(int i = 0; i < pathsize ;i++){
            //finds the first one that wasn't touched
            if( *(ptr_path + (i*vertices))  == -1){
                //maybe there could be a way to binary search this to make it faster!
                //i dunno :3
                //paths[i] = ha;
                for(int j =0; j < vertices ; j++){
                    //write to the original array
                    *(ptr_path + (i*vertices)+j) = ha[j];
                }
                //stops when found
                return;
            }
        }
}


void permute(int str[] , int l, int r, int *ptr_path, int vertices, int pathsize){
        //printf("Value of paths = %p\n",ptr_path);
        if (l == r){
            //array created to save each combination
            int raw [vertices]; 
            //prints the combinations and save them to the array.
            for(int j=0; j < vertices ;j++){
                //printf("%i", str[j]);
                raw[j] = str[j];
            }
           getArray(raw,ptr_path,vertices,pathsize);
           //printf("+++\n");
            //count = 0;
        }
        else
        {
            for (int i = l; i <= r; i++)
            {
                str[i] = swap(str,l,i);
                //permute(str, l+1, r);
                permute(str, l+1, r,ptr_path, vertices, pathsize);
                str[i] = swap(str,l,i);    
            }
        }
        
    }





int main(void) {

	const int vertices = 5;

	int *matrix;
	//int matrix[vertices*vertices] ;
 	//memset(matrix, 0, sizeof matrix);
	
	//1D MATRIX ARRAY INNITIALLIZED IN 
	//ALLOCATES
	hipMallocManaged(&matrix, (vertices*vertices)*sizeof(int));
	//initiates to 0
	initTo<<<1,100>>>(matrix,vertices*vertices,0);
	hipDeviceSynchronize();


	int pathsize = 1;
    	for (int i = 1; i < vertices + 1; i++)
    	{
        	pathsize = pathsize * i;
    	}
    	printf("%i\n",pathsize);

	bool *possiblePaths;
	hipMallocManaged(&possiblePaths, (pathsize)*sizeof(bool));

	//bool possiblePaths[pathsize];
	//initialize to true
    	//memset(possiblePaths, true, sizeof possiblePaths);
	initTo<<<1,100>>>(possiblePaths,pathsize,true);
	hipDeviceSynchronize();
	
	int *allPaths;
	hipMallocManaged(&allPaths, (pathsize*vertices)*sizeof(int));
	initTo<<<1,100>>>(allPaths,pathsize*vertices,-1);
	hipDeviceSynchronize();



//creating the graph 
	// ---
	//| / |
	// ---

	
	makeEdge(0,2,matrix,vertices);
	makeEdge(0,3,matrix,vertices);
   	makeEdge(1,4,matrix,vertices);
    	makeEdge(1,3,matrix,vertices);
	makeEdge(2,4,matrix,vertices);

	printf("The adjacency matrix for the given graph is: ");
   	 printf("\n  ");
    	for (int i = 0; i < vertices; i++)
       		printf("%i ",i+1);
    
    	for (int j = 0; j < vertices*vertices; j++) {
        	if(j % vertices == 0)
                	printf("\n%i ",(j/4)+1);
            printf("%i ", matrix[j]);
    	}
        printf("\n");

printf("%i\n",allPaths[0]);
printf("%i\n",allPaths[95]);	

int arr[vertices];
	for(int pattern = 0 ; pattern < vertices; pattern++){
        	arr[pattern] = pattern;
}

permute(arr,0,vertices - 1,allPaths,vertices,pathsize);

//printf("%i\n",allPaths[0]);
//printf("%i\n",allPaths[95]);
for (int i = 0; i < pathsize; i++)
        {
            if(i<10)
            printf(" %i |",i);
            else
                printf("%i |",i);

            for (int j = 0; j < vertices; j++)
            {
                printf("%d ", allPaths[(i*vertices)+j]);
            }
            //ptr_path++;
            printf("\n");
		
        }
/*
printf("%i\n",allPaths[92]);
printf("%i\n",allPaths[93]);
printf("%i\n",allPaths[94]);
printf("%i\n",allPaths[95]);
*/


starttime();
checkPathLocal(allPaths,vertices,pathsize,matrix,possiblePaths);
hipDeviceSynchronize();

printf("POSSIBLE PATHS\n");
for(int wow = 0 ; wow<pathsize;wow++){
        //printf("");
        if(possiblePaths[wow] == true){
            for(int innerwow = 0 ; innerwow < vertices ; innerwow++){
                printf("%i", allPaths[(wow*vertices)+innerwow]);
            }
            printf("\n");
        }
        //printf("%d ",possiblePaths[wow]);
    }
printf("\n");

hipDeviceSynchronize();
initTo<<<1,100>>>(possiblePaths,pathsize,true);
hipDeviceSynchronize();

hipDeviceSynchronize();
starttime();
checkPathsTH<<<2,75>>>(matrix,vertices,pathsize,allPaths,possiblePaths);
endtime("GPU THREADS");
hipDeviceSynchronize();

hipDeviceSynchronize();
printf("POSSIBLE PATHS\n");
for(int wow = 0 ; wow<pathsize;wow++){
        //printf("");
        if(possiblePaths[wow] == true){
            for(int innerwow = 0 ; innerwow < vertices ; innerwow++){
                printf("%i", allPaths[(wow*vertices)+innerwow]);
            }
            printf("\n");
        }
        //printf("%d ",possiblePaths[wow]);
    }
printf("\n");




	 // Free memory
  	hipFree(matrix);
	hipFree(possiblePaths);
	hipFree(allPaths);
	//free(matrix);

	return 0;
}